#include "hip/hip_runtime.h"
// BSD 3-Clause License

// Copyright (c) 2021, Muhammad Asad (masadcv@gmail.com)
// All rights reserved.

// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:

// 1. Redistributions of source code must retain the above copyright notice, this
//    list of conditions and the following disclaimer.

// 2. Redistributions in binary form must reproduce the above copyright notice,
//    this list of conditions and the following disclaimer in the documentation
//    and/or other materials provided with the distribution.

// 3. Neither the name of the copyright holder nor the names of its
//    contributors may be used to endorse or promote products derived from
//    this software without specific prior written permission.

// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
// AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
// DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
// SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
// CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
// OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <iostream>
#ifdef _OPENMP
#include <omp.h>
#endif

#define THREAD_COUNT 256

// whether to use float* or Pytorch accessors in CUDA kernels
#define USE_PTR 1

__constant__ float local_dist2d[3];
__constant__ float local_dist3d[3 * 3];

__device__ float l1distance_cuda(const float &in1, const float &in2)
{
    return abs(in1 - in2);
}

template <typename scalar_t>
__global__ void geodesic_updown_single_row_pass_kernel(
    torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> image_ptr,
    torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> distance_ptr,
    const float l_grad,
    const float l_eucl,
    const int direction)
{
    const int channel = image_ptr.size(1);
    const int height = image_ptr.size(2);
    const int width = image_ptr.size(3);

    int kernelW = blockIdx.x * blockDim.x + threadIdx.x;

    int h = (direction == 1)? 1 : height - 2; 

    // if outside, then skip distance calculation - dont use the thread
    if (kernelW < width)
    {
        while (h >= 0 && h < height)
        {
            int prevH = h - direction;
            if (prevH < 0 || prevH >= height)
            {
                // read outside bounds, skip
                continue;
            }
            
            float l_dist, cur_dist;
            float new_dist = distance_ptr[0][0][h][kernelW];

            for (int w_i = 0; w_i < 3; w_i++)
            {
                const int kernelW_ind = kernelW + w_i - 1;

                if (kernelW_ind >= 0 && kernelW_ind < width)
                {
                    l_dist = 0.0;
                    if (channel == 1)
                    {
                        l_dist = l1distance_cuda(
                            image_ptr[0][0][h][kernelW], 
                            image_ptr[0][0][prevH][kernelW_ind]
                            );
                    }
                    else
                    {
                        for (int c_i = 0; c_i < channel; c_i++)
                        {
                            l_dist += l1distance_cuda(
                                image_ptr[0][c_i][h][kernelW], 
                                image_ptr[0][c_i][prevH][kernelW_ind]
                                );
                        }
                    }
                    cur_dist = distance_ptr[0][0][prevH][kernelW_ind] + \
                                l_eucl * local_dist2d[w_i] + \
                                l_grad * l_dist;

                    new_dist = std::min(new_dist, cur_dist);
                }
            }
            if (new_dist < distance_ptr[0][0][h][kernelW])
            {
                distance_ptr[0][0][h][kernelW] = new_dist;
            }

            // go to next row
            h += direction;

            // synchronise write for all threads
            __syncthreads();
        }
    }
}

__global__ void geodesic_updown_single_row_pass_ptr_kernel(
    float *image_ptr,
    float *distance_ptr,
    const float l_grad,
    const float l_eucl,
    const int direction,
    const int channel,
    const int height,
    const int width)
{
    int kernelW = blockIdx.x * blockDim.x + threadIdx.x;

    int h = (direction == 1)? 1 : height - 2; 

    // if outside, then skip distance calculation - dont use the thread
    if (kernelW < width)
    {
        while (h >= 0 && h < height)
        {
            int prevH = h - direction;
            if (prevH < 0 || prevH >= height)
            {
                // read outside bounds, skip
                continue;
            }
            float l_dist, cur_dist;
            float new_dist = distance_ptr[h * width + kernelW];

            for (int w_i = 0; w_i < 3; w_i++)
            {
                const int kernelW_ind = kernelW + w_i - 1;

                if (kernelW_ind >= 0 && kernelW_ind < width)
                {
                    l_dist = 0.0;
                    if (channel == 1)
                    {
                        l_dist = l1distance_cuda(
                            image_ptr[h * width + kernelW], 
                            image_ptr[(prevH) * width + kernelW_ind]
                            );
                    }
                    else
                    {
                        for (int c_i = 0; c_i < channel; c_i++)
                        {
                            l_dist += l1distance_cuda(
                                image_ptr[c_i * height * width + h * width + kernelW], 
                                image_ptr[c_i * height * width + (prevH) * width + kernelW_ind]
                                );
                        }
                    }
                    cur_dist = distance_ptr[(prevH) * width + kernelW_ind] + \
                                l_eucl * local_dist2d[w_i] + \
                                l_grad * l_dist;

                    new_dist = std::min(new_dist, cur_dist);
                }
            }
            if (new_dist < distance_ptr[h * width + kernelW])
            {
                distance_ptr[h * width + kernelW] = new_dist;
            }

            // go to next row
            h += direction;

            // synchronise write for all threads
            __syncthreads();
        }
    }
}

void geodesic_updown_pass_cuda(
    const torch::Tensor image, 
    torch::Tensor distance, 
    const float &l_grad, 
    const float &l_eucl
    )
{
    // batch, channel, height, width
    const int channel = image.size(1);
    const int height = image.size(2);
    const int width = image.size(3);

    // constexpr float local_dist[] = {sqrt(2.), 1., sqrt(2.)};
    const float local_dist[] = {sqrt(float(2.)), float(1.), sqrt(float(2.))};

    // copy local distances to GPU __constant__ memory
    hipMemcpyToSymbol(HIP_SYMBOL(local_dist2d), local_dist, sizeof(float) * 3);

    int blockCountUpDown = (width + 1) / THREAD_COUNT + 1;

    // direction variable used to indicate read from previous (-1) or next (+1) row
    int direction;

    // top-down
    direction = +1;
    // each distance calculation in down pass require previous row i.e. +1
    // process each row in parallel with CUDA kernel
    #if USE_PTR
        geodesic_updown_single_row_pass_ptr_kernel<<<blockCountUpDown, THREAD_COUNT>>>(
            image.data_ptr<float>(),
            distance.data_ptr<float>(),
            l_grad,
            l_eucl,
            direction,
            channel,
            height,
            width);
    #else
        AT_DISPATCH_FLOATING_TYPES(image.type(), "geodesic_updown_single_row_pass_kernel", ([&]
            { geodesic_updown_single_row_pass_kernel<scalar_t><<<blockCountUpDown, THREAD_COUNT>>>(
                image.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
                distance.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
                l_grad,
                l_eucl,
                direction); 
            }));
    #endif


    // bottom-up
    direction = -1;
    #if USE_PTR
        geodesic_updown_single_row_pass_ptr_kernel<<<blockCountUpDown, THREAD_COUNT>>>(
            image.data_ptr<float>(),
            distance.data_ptr<float>(),
            l_grad,
            l_eucl,
            direction,
            channel,
            height,
            width);
    #else
        AT_DISPATCH_FLOATING_TYPES(image.type(), "geodesic_updown_single_row_pass_kernel", ([&]
            { geodesic_updown_single_row_pass_kernel<scalar_t><<<blockCountUpDown, THREAD_COUNT>>>(
                image.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
                distance.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>(),
                l_grad,
                l_eucl,
                direction); 
            }));
    #endif
    
}

torch::Tensor generalised_geodesic2d_cuda(
    const torch::Tensor &image, 
    const torch::Tensor &mask, 
    const float &v, 
    const float &l_grad, 
    const float &l_eucl, 
    const int &iterations
    )
{
    int device = image.get_device();
    // std::cout << "Running with CUDA Device: " << device << std::endl;
    c10::cuda::CUDAGuard device_guard(device);
    
    torch::Tensor image_local = image.clone();
    torch::Tensor distance = v * mask.clone();

    // iteratively run the distance transform
    for (int itr = 0; itr < iterations; itr++)
    {
        image_local = image_local.contiguous();
        distance = distance.contiguous();

        // top-bottom - width*, height
        geodesic_updown_pass_cuda(image_local, distance, l_grad, l_eucl);

        // left-right - height*, width
        image_local = image_local.transpose(2, 3);
        distance = distance.transpose(2, 3);

        image_local = image_local.contiguous();
        distance = distance.contiguous();
        geodesic_updown_pass_cuda(image_local, distance, l_grad, l_eucl);

        // tranpose back to original - width, height
        image_local = image_local.transpose(2, 3);
        distance = distance.transpose(2, 3);

        // * indicates the current direction of pass
    }
    return distance;
}


torch::Tensor pairwise_geodesic2d_cuda(
    const torch::Tensor &image,
    const float &v,
    const float &l_grad,
    const float &l_eucl,
    const int &iterations)
{
    int device = image.get_device();
    c10::cuda::CUDAGuard device_guard(device);

    const int height = image.size(2);
    const int width = image.size(3);
    const int num_pixels = height * width;

    // 预分配最终的成对距离矩阵
    auto options = image.options();
    torch::Tensor pairwise_matrix = torch::zeros({num_pixels, num_pixels}, options);

    // 创建一个掩码张量，将当前像素设置为唯一的种子点
    torch::Tensor mask = torch::ones({1,1,height, width}, options);

    // 为每个像素作为种子点进行迭代
    for (int i = 0; i < num_pixels; ++i)
    {
        // 将一维索引转换为二维坐标
        int seed_h = i / width;
        int seed_w = i % width;

        // 设置种子点值为0，其他为1
        mask.index_put_({0, 0, seed_h, seed_w}, 0.0);

        //torch::Tensor single_source_distances = torch::zeros({1,1,height, width}, options);
        torch::Tensor single_source_distances = generalised_geodesic2d_cuda(image, mask, v, l_grad, l_eucl, iterations);

        // 将 HxW 的距离图展平为 1D 向量，并存入结果矩阵的对应行
        pairwise_matrix.select(0, i).copy_(single_source_distances.flatten());
        mask.index_put_({0, 0, seed_h, seed_w}, 1.0);
    }

    return pairwise_matrix;
}

template <typename scalar_t>
__global__ void geodesic_frontback_single_plane_pass_kernel(
    torch::PackedTensorAccessor32<scalar_t, 5, torch::RestrictPtrTraits> image_ptr,
    torch::PackedTensorAccessor32<scalar_t, 5, torch::RestrictPtrTraits> distance_ptr,
    const float l_grad,
    const float l_eucl,
    const int direction)
{
    const int channel = image_ptr.size(1);
    const int depth = image_ptr.size(2);
    const int height = image_ptr.size(3);
    const int width = image_ptr.size(4);

    int kernelW = blockIdx.x * blockDim.x + threadIdx.x;
    int kernelH = blockIdx.y * blockDim.y + threadIdx.y;

    int z = (direction == 1) ? 1 : depth - 2;

    // if outside, then skip distance calculation - dont use the thread
    if (kernelH >= 0 && kernelH < height && kernelW >= 0 && kernelW < width)
    {
        while (z >= 0 && z < depth)
        {
            int prevZ = z - direction;
            if (prevZ < 0 || prevZ >= depth)
            {
                // go to next plane
                z += direction;

                // read outside bounds, skip
                continue;
            }
            float l_dist, cur_dist;
            float new_dist = distance_ptr[0][0][z][kernelH][kernelW];

            for (int h_i = 0; h_i < 3; h_i++)
            {
                for (int w_i = 0; w_i < 3; w_i++)
                {
                    const int kernelH_ind = kernelH + h_i - 1;
                    const int kernelW_ind = kernelW + w_i - 1;

                    if (kernelH_ind >= 0 && \
                        kernelH_ind < height && \
                        kernelW_ind >= 0 && \
                        kernelW_ind < width)
                    {
                        l_dist = 0.0;
                        if (channel == 1)
                        {
                            l_dist = l1distance_cuda(
                                        image_ptr[0][0][z][kernelH][kernelW], 
                                        image_ptr[0][0][prevZ][kernelH_ind][kernelW_ind]
                                        );
                        }
                        else
                        {
                            for (int c_i = 0; c_i < channel; c_i++)
                            {
                                l_dist += l1distance_cuda(
                                    image_ptr[0][c_i][z][kernelH][kernelW], 
                                    image_ptr[0][c_i][prevZ][kernelH_ind][kernelW_ind]
                                    );
                            }
                        }
                        cur_dist = distance_ptr[0][0][prevZ][kernelH_ind][kernelW_ind] + \
                                    l_eucl * local_dist3d[h_i * 3 + w_i] + \
                                    l_grad * l_dist;

                        new_dist = std::min(new_dist, cur_dist);
                    }
                }
            }
            if (new_dist < distance_ptr[0][0][z][kernelH][kernelW])
            {
                distance_ptr[0][0][z][kernelH][kernelW] = new_dist;
            }

            // go to next plane
            z += direction;

            // synchronise write for all threads
            __syncthreads();
        }
    }
}

__global__ void geodesic_frontback_single_plane_pass_ptr_kernel(
    float *image_ptr,
    float *distance_ptr,
    const float l_grad,
    const float l_eucl,
    const int direction,
    const int channel,
    const int depth,
    const int height,
    const int width
    )
{

    int kernelW = blockIdx.x * blockDim.x + threadIdx.x;
    int kernelH = blockIdx.y * blockDim.y + threadIdx.y;

    int z = (direction == 1) ? 1 : depth - 2;

    // if outside, then skip distance calculation - dont use the thread
    if (kernelH >= 0 && \
        kernelH < height && \
        kernelW >= 0 && \
        kernelW < width)
    {
        while (z >= 0 && z < depth)
        {
            int prevZ = z - direction;
            if (prevZ < 0 || prevZ >= depth)
            {
                // go to next plane
                z += direction;
                
                // read outside bounds, skip
                continue;
            }
            float l_dist, cur_dist;
            float new_dist = distance_ptr[z * height * width + kernelH * width + kernelW];

            for (int h_i = 0; h_i < 3; h_i++)
            {
                for (int w_i = 0; w_i < 3; w_i++)
                {
                    const int kernelH_ind = kernelH + h_i - 1;
                    const int kernelW_ind = kernelW + w_i - 1;

                    if (kernelH_ind >= 0 && \
                        kernelH_ind < height && \
                        kernelW_ind >= 0 && \
                        kernelW_ind < width)
                    {
                        l_dist = 0.0;
                        if (channel == 1)
                        {
                            l_dist = l1distance_cuda(
                                        image_ptr[z * height * width + kernelH * width + kernelW], 
                                        image_ptr[prevZ * height * width + kernelH_ind * width + kernelW_ind]
                                        );
                        }
                        else
                        {
                            for (int c_i = 0; c_i < channel; c_i++)
                            {
                                l_dist += l1distance_cuda(
                                            image_ptr[c_i * depth * height * width + z * height * width + kernelH * width + kernelW], 
                                            image_ptr[c_i * depth * height * width + prevZ * height * width + kernelH_ind * width + kernelW_ind]);
                            }
                        }
                        cur_dist = distance_ptr[prevZ * height * width + kernelH_ind * width + kernelW_ind] + \
                                    l_eucl * local_dist3d[h_i * 3 + w_i] + \
                                    l_grad * l_dist;
                        new_dist = std::min(new_dist, cur_dist);
                    }
                }
            }
            if (new_dist < distance_ptr[z * height * width + kernelH * width + kernelW])
            {
                distance_ptr[z * height * width + kernelH * width + kernelW] = new_dist;
            }

            // go to next plane
            z += direction;

            // synchronise write for all threads
            __syncthreads();
        }
    }
}

void geodesic_frontback_pass_cuda(
    const torch::Tensor &image, 
    torch::Tensor &distance, 
    const std::vector<float> &spacing, 
    const float &l_grad, 
    const float &l_eucl
    )
{
    // batch, channel, depth, height, width
    const int channel = image.size(1);
    const int depth = image.size(2);
    const int height = image.size(3);
    const int width = image.size(4);

    // convert allowed number of threads into a 2D grid
    // helps if the THREAD_COUNT is N*N already
    const int THREAD_COUNT_2D = sqrt(THREAD_COUNT);
    int blockCountUpDown = (width + 1) / THREAD_COUNT_2D + 1;
    int blockCountLeftRight = (height + 1) / THREAD_COUNT_2D + 1;

    // pre-calculate local distances based on spacing
    float local_dist[3 * 3];
    for (int h_i = 0; h_i < 3; h_i++)
    {
        for (int w_i = 0; w_i < 3; w_i++)
        {
            float ld = spacing[0];
            ld += float(std::abs(h_i - 1)) * spacing[1];
            ld += float(std::abs(w_i - 1)) * spacing[2];

            local_dist[h_i * 3 + w_i] = sqrt(ld);
        }
    }
    // copy local distances to GPU __constant__ memory
    hipMemcpyToSymbol(HIP_SYMBOL(local_dist3d), local_dist, sizeof(float) * 3 * 3);

    dim3 dimGrid(blockCountUpDown, blockCountLeftRight);
    dim3 dimBlock(THREAD_COUNT_2D, THREAD_COUNT_2D);
    // Kernel<<<dimGrid, dimBlock>>>( arg1, arg2, arg2);

    // direction variable used to indicate read from previous (+1) or next (-1) plane
    int direction;

    // front-back
    direction = +1;
    #if USE_PTR
        geodesic_frontback_single_plane_pass_ptr_kernel<<<dimGrid, dimBlock>>>(
            image.data_ptr<float>(),
            distance.data_ptr<float>(),
            l_grad,
            l_eucl,
            direction,
            channel,
            depth,
            height,
            width);
    #else
        AT_DISPATCH_FLOATING_TYPES(image.type(), "geodesic_frontback_single_plane_pass_kernel", ([&]
            { geodesic_frontback_single_plane_pass_kernel<scalar_t><<<dimGrid, dimBlock>>>(
                image.packed_accessor32<scalar_t, 5, torch::RestrictPtrTraits>(),
                distance.packed_accessor32<scalar_t, 5, torch::RestrictPtrTraits>(),
                l_grad,
                l_eucl,
                direction); 
            }));
    #endif

    // back-front
    direction = -1;
    #if USE_PTR
        geodesic_frontback_single_plane_pass_ptr_kernel<<<dimGrid, dimBlock>>>(
            image.data_ptr<float>(),
            distance.data_ptr<float>(),
            l_grad,
            l_eucl,
            direction,
            channel,
            depth,
            height,
            width);
    #else
        AT_DISPATCH_FLOATING_TYPES(image.type(), "geodesic_frontback_single_plane_pass_kernel", ([&]
            { geodesic_frontback_single_plane_pass_kernel<scalar_t><<<dimGrid, dimBlock>>>(
                image.packed_accessor32<scalar_t, 5, torch::RestrictPtrTraits>(),
                distance.packed_accessor32<scalar_t, 5, torch::RestrictPtrTraits>(),
                l_grad,
                l_eucl,
                direction); 
            }));
    #endif
}

torch::Tensor generalised_geodesic3d_cuda(
    const torch::Tensor &image, 
    const torch::Tensor &mask, 
    std::vector<float> spacing, 
    const float &v, 
    const float &l_grad, 
    const float &l_eucl, 
    const int &iterations
    )
{
    int device = image.get_device();
    // std::cout << "Running with CUDA Device: " << device << std::endl;
    c10::cuda::CUDAGuard device_guard(device);

    // square spacing with transform
    std::transform(spacing.begin(), spacing.end(), spacing.begin(), spacing.begin(), std::multiplies<float>());
    
    torch::Tensor image_local = image.clone();
    torch::Tensor distance = v * mask.clone();

    // iteratively run the distance transform
    for (int itr = 0; itr < iterations; itr++)
    {
        image_local = image_local.contiguous();
        distance = distance.contiguous();

        // front-back - depth*, height, width
        geodesic_frontback_pass_cuda(image_local, distance, spacing, l_grad, l_eucl);

        // top-bottom - height*, depth, width
        image_local = torch::transpose(image_local, 3, 2);
        distance = torch::transpose(distance, 3, 2);

        image_local = image_local.contiguous();
        distance = distance.contiguous();
        geodesic_frontback_pass_cuda(
            image_local,
            distance,
            {spacing[1], spacing[0], spacing[2]},
            l_grad,
            l_eucl
            );

        // transpose back to original depth, height, width
        image_local = torch::transpose(image_local, 3, 2);
        distance = torch::transpose(distance, 3, 2);

        // left-right - width*, height, depth
        image_local = torch::transpose(image_local, 4, 2);
        distance = torch::transpose(distance, 4, 2);

        image_local = image_local.contiguous();
        distance = distance.contiguous();
        geodesic_frontback_pass_cuda(
            image_local,
            distance,
            {spacing[2], spacing[1], spacing[0]},
            l_grad,
            l_eucl
            );

        // transpose back to original depth, height, width
        image_local = torch::transpose(image_local, 4, 2);
        distance = torch::transpose(distance, 4, 2);

        // * indicates the current direction of pass
    }

    return distance;
}

// image: (1, C, H, W) float32 contiguous
// distance: (1,1,H,W) float32
// low/high: (C,H,W) float32 buffers (per-channel)
// direction: +1 for top->bottom, -1 for bottom->top
__global__ void mbd_updown_single_row_pass_ptr_kernel(
    const float* __restrict__ image_ptr,
    float* __restrict__ distance_ptr,
    float* __restrict__ low_ptr,
    float* __restrict__ high_ptr,
    const int direction,
    const int channel,
    const int height,
    const int width
){
    int w = blockIdx.x * blockDim.x + threadIdx.x;
    int h = (direction == 1)? 1 : height - 2;

    if (w >= width) return;

    while (h >= 0 && h < height) {
        int ph = h - direction; // previous row index in the scanning direction
        if (ph < 0 || ph >= height) {
            // advance to next row if out of bounds
            h += direction;
            __syncthreads();
            continue;
        }

        // Current best values at (h,w)
        int cur_idx = h * width + w;
        float best_cost = distance_ptr[cur_idx];

        // Load current per-channel low/high as fallback (if no improvement)
        // Note: low/high arrays are laid out [C, H, W]
        // idx(c,h,w) = c*H*W + h*W + w
        // We'll keep temporary best low/high in registers (per-channel)
        // If channel is large, we loop twice: once to compute candidate, once to commit.
        // Here we store only when improving to avoid extra memory traffic.
        // We will compute candidates per neighbor and keep the best seen.

        // Variables to remember which neighbor produced the best candidate
        int best_neighbor_w = -999; // -1,0,+1 offset encoded by absolute w index
        // We won't store per-channel best L/U here to save registers; we recompute once when applying.

        // Try 3 neighbors from the previous row: (ph, w-1), (ph, w), (ph, w+1)
        for (int wi = -1; wi <= 1; ++wi) {
            int nw = w + wi;
            if (nw < 0 || nw >= width) continue;

            int n_idx = ph * width + nw;

            // For candidate cost, we need to combine neighbor's low/high with current pixel intensity
            // Compute D' = sum_c ( max(U_n[c], I[h,w,c]) - min(L_n[c], I[h,w,c]) )
            float cand_cost = 0.f;

            for (int c = 0; c < channel; ++c) {
                int off_cur = c * (height * width) + cur_idx;
                int off_nei = c * (height * width) + n_idx;
                float Icur = image_ptr[off_cur];
                float Lnei = low_ptr[off_nei];
                float Unei = high_ptr[off_nei];
                float Lc = fminf(Lnei, Icur);
                float Uc = fmaxf(Unei, Icur);
                cand_cost += (Uc - Lc);
            }

            if (cand_cost < best_cost) {
                best_cost = cand_cost;
                best_neighbor_w = nw;
            }
        }

        if (best_cost < distance_ptr[cur_idx]) {
            // Commit update: write distance, and update low/high at (h,w) using the neighbor that gave best_cost
            distance_ptr[cur_idx] = best_cost;

            int n_idx = ph * width + best_neighbor_w;
            for (int c = 0; c < channel; ++c) {
                int off_cur = c * (height * width) + cur_idx;
                int off_nei = c * (height * width) + n_idx;
                float Icur = image_ptr[off_cur];
                float Lnei = low_ptr[off_nei];
                float Unei = high_ptr[off_nei];
                float Lc = fminf(Lnei, Icur);
                float Uc = fmaxf(Unei, Icur);
                low_ptr[off_cur]  = Lc;
                high_ptr[off_cur] = Uc;
            }
        }

        h += direction;
        __syncthreads();
    }
}

static inline void mbd_updown_pass_cuda(
    const torch::Tensor& image,
    torch::Tensor& distance,
    torch::Tensor& low,
    torch::Tensor& high
){
    const int channel = image.size(1);
    const int height  = image.size(2);
    const int width   = image.size(3);

    int blockCount = (width + THREAD_COUNT - 1) / THREAD_COUNT;

    // top -> bottom
    mbd_updown_single_row_pass_ptr_kernel<<<blockCount, THREAD_COUNT>>>(
        image.data_ptr<float>(),
        distance.data_ptr<float>(),
        low.data_ptr<float>(),
        high.data_ptr<float>(),
        +1, channel, height, width
    );

    // bottom -> top
    mbd_updown_single_row_pass_ptr_kernel<<<blockCount, THREAD_COUNT>>>(
        image.data_ptr<float>(),
        distance.data_ptr<float>(),
        low.data_ptr<float>(),
        high.data_ptr<float>(),
        -1, channel, height, width
    );
}

// Public API: single-source MBD with given seeds mask (0 at seeds, 1 elsewhere)
// Returns (1,1,H,W) distance tensor
torch::Tensor mbd2d_cuda(
    const torch::Tensor& image,   // (1,C,H,W) float32, on CUDA
    const torch::Tensor& mask,    // (1,1,H,W) float32, 0 at seeds, 1 elsewhere
    const float& v,               // initial large value, e.g., 1e6
    const int& iterations         // number of full 4-direction iterations
){
    TORCH_CHECK(image.is_cuda(), "image must be CUDA tensor");
    TORCH_CHECK(mask.is_cuda(), "mask must be CUDA tensor");
    TORCH_CHECK(image.dim() == 4 && image.size(0) == 1, "image must be (1,C,H,W)");
    TORCH_CHECK(mask.dim() == 4 && mask.sizes() == torch::IntArrayRef({1,1,image.size(2), image.size(3)}),
                "mask must be (1,1,H,W) matching image");

    c10::cuda::CUDAGuard device_guard(image.get_device());
    auto optsF = image.options().dtype(torch::kFloat32);

    const int C = image.size(1);
    const int H = image.size(2);
    const int W = image.size(3);

    // Distance init: seeds=0, others=v
    torch::Tensor distance = v * mask.clone();

    // Low/High init: start with per-pixel intensity (so extending a path simply tightens/loosens)
    // Shape (C,H,W)
    torch::Tensor img_chw = image[0]; // (C,H,W)
    torch::Tensor low  = img_chw.clone();
    torch::Tensor high = img_chw.clone();

    for (int itr = 0; itr < iterations; ++itr) {
        // Vertical passes
        mbd_updown_pass_cuda(image, distance, low, high);

        // Transpose to reuse the same kernel as "vertical" over width
        // We need to transpose image (C,H,W) -> (C,W,H) and distance/low/high accordingly.
        // Work in (1,C,H,W) for image and (C,H,W) for low/high to keep pointers consistent.
        torch::Tensor image_t   = image.transpose(2, 3).contiguous();
        torch::Tensor distance_t= distance.transpose(2, 3).contiguous();
        torch::Tensor low_t     = low.transpose(1, 2).contiguous();   // (C,W,H)
        torch::Tensor high_t    = high.transpose(1, 2).contiguous();  // (C,W,H)

        mbd_updown_pass_cuda(image_t, distance_t, low_t, high_t);

        // transpose back
        distance = distance_t.transpose(2, 3).contiguous();
        low      = low_t.transpose(1, 2).contiguous();
        high     = high_t.transpose(1, 2).contiguous();
    }

    return distance;
}